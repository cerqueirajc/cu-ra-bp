#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define PI 3.141592
#define ini -0.7071067811865475244008
#define b_TPBX 16
#define b_TPBY 16

texture<float, hipTextureType2D, hipReadModeElementType> texRefSino;


#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }


__global__ void backp_kernel(
		float *image,
		int wdI,
		int nrays,
		int nangles,
		float delta,
		float dt,
		float dth)

{
  int i, j, T;
  float t, cumsum, k;
  float	x;
  float y;

  i = blockDim.x * blockIdx.x + threadIdx.x;
  j = blockDim.y * blockIdx.y + threadIdx.y;

  if ((i<wdI) && (j < wdI) ){

  x = (float)ini + i * delta;
  y = (float)ini + j * delta;


  cumsum = 0;

  for(k=0; k < nangles; k++)
    {

      t = x*cosf(k * dth) + y*sinf(k * dth);

      T = (float)((t + 1)/dt);

      if(T > -1 && T < nrays){
	  cumsum = cumsum + tex2D(texRefSino, k + 0.5f, T + 0.5f);
      }
    }
  image[j*wdI + i] = (cumsum*dth);
}
}

__global__ void ray_sum_kernel(float *a, float *b, size_t size, size_t size2, size_t size3)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) {
        return;
    }

    b[idx] = 2*a[idx];
}


extern "C" {
void Back(float *image, float *sino, int sizeImage, int nrays, int nangles){
    float* d_output;
	int size = nrays * nangles * sizeof(float);
	float dt  = 2.0/(nrays-1);
	float dth = PI/(nangles);
	float delta = (double) sqrt(2)/(sizeImage-1);


			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			float milliseconds = 0;

			hipEventRecord(start);

    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, nangles, nrays);

    // Copy to device memory some data located at address h_data in host memory
    hipMemcpyToArray(cuArray, 0, 0, sino, size , hipMemcpyHostToDevice);

    // Set texture parameters
    texRefSino.addressMode[0] = hipAddressModeWrap;
    texRefSino.addressMode[1] = hipAddressModeWrap;
    texRefSino.filterMode     = hipFilterModeLinear;
  //texRefSino.normalized     = true;

    // Bind the array to the texture reference
    hipBindTextureToArray(texRefSino, cuArray, channelDesc);

    // Allocate GPU buffers for the output image ..
    hipMalloc(&d_output, sizeof(float) * sizeImage *sizeImage);

    //GRID N BLOCKS SIZE

    dim3 threadsPerBlock(b_TPBX,b_TPBY);
    dim3 grid((sizeImage/threadsPerBlock.x) + 1, (sizeImage/threadsPerBlock.y) + 1);


    backp_kernel<<<grid, threadsPerBlock>>>(d_output, sizeImage, nrays, nangles, delta, dt, dth);

	hipGetLastError();

				hipEventRecord(stop);

				hipEventSynchronize(stop);
				hipEventElapsedTime(&milliseconds, start, stop);

				fprintf(stderr, "%f ms\n", milliseconds);

	hipMemcpy (image , d_output , sizeImage*sizeImage*sizeof(float) , hipMemcpyDeviceToHost);

    hipUnbindTexture(texRefSino);
    hipFreeArray(cuArray);
    hipFree(d_output);
    //hipDeviceReset();
}}


extern "C" {
void raft_backprojection_slantstack_gpu(float *a, float *b, size_t size, size_t size2, size_t size3)
{
    float *d_a, *d_b;

    hipMalloc((void **)&d_a, size * sizeof(float));
    hipMalloc((void **)&d_b, size * sizeof(float));


    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    ray_sum_kernel <<< ceil(size / 256.0), 256 >>> (d_a, d_b, size, size2, size3);

    hipMemcpy(b, d_b, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);

}
}

