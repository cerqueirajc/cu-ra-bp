#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define PI 3.141592
#define ini -0.7071067811865475244008
#define TPBX 16
#define TPBY 16

texture<float, hipTextureType2D, hipReadModeElementType> texRefSino;

#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }


__global__ void backp_kernel(
		float *image,
		int wdI,
		int nrays,
		int nangles,
		float delta,
		float dt,
		float dth)

{
  int i, j, T;
  float t, cumsum, k;
  float	x;
  float y;

  i = blockDim.x * blockIdx.x + threadIdx.x;
  j = blockDim.y * blockIdx.y + threadIdx.y;

  if ((i<wdI) && (j < wdI) ){

  x = (float)ini + i * delta;
  y = (float)ini + j * delta;


  cumsum = 0;

	#pragma unroll 1000
  for(k=0; k < nangles; k++)
    {

      t = x*cosf(k * dth) + y*sinf(k * dth);

      T = (float)((t + 1)/dt);

      if(T > -1 && T < nrays){
	  cumsum = cumsum + tex2D(texRefSino, k + 0.5f, T + 0.5f);
      }
    }
  image[j*wdI + i] = (cumsum*dth);
}
}


void BackWithTexture(float *image, float *sino, int sizeImage, int nrays, int nangles){
    float* d_output;
	int size = nrays * nangles * sizeof(float);
	float dt  = 2.0/(nrays-1);
	float dth = PI/(nangles);
	float delta = (double) sqrt(2)/(sizeImage-1);


			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			float milliseconds = 0;
			hipEventRecord(start);

    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, nangles, nrays);

    // Copy to device memory some data located at address h_data in host memory
    hipMemcpyToArray(cuArray, 0, 0, sino, size , hipMemcpyHostToDevice);

    // Set texture parameters
    texRefSino.addressMode[0] = hipAddressModeWrap;
    texRefSino.addressMode[1] = hipAddressModeWrap;
    texRefSino.filterMode     = hipFilterModeLinear;
  //texRefSino.normalized     = true;

    // Bind the array to the texture reference
    hipBindTextureToArray(texRefSino, cuArray, channelDesc);

    // Allocate GPU buffers for the output image ..
    hipMalloc(&d_output, sizeof(float) * sizeImage *sizeImage);

    //GRID N BLOCKS SIZE

    dim3 threadsPerBlock(TPBX,TPBY);
    dim3 grid((sizeImage/threadsPerBlock.x) + 1, (sizeImage/threadsPerBlock.y) + 1);


    backp_kernel<<<grid, threadsPerBlock>>>(d_output, sizeImage, nrays, nangles, delta, dt, dth);

	hipGetLastError();
		
	hipMemcpy (image , d_output , sizeImage*sizeImage*sizeof(float) , hipMemcpyDeviceToHost);

				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&milliseconds, start, stop);

				fprintf(stderr, "%f ms\n", milliseconds);

	

    hipUnbindTexture(texRefSino);
    hipFreeArray(cuArray);
    hipFree(d_output);
    hipDeviceReset();
}



int main(int argc, char *argv[]) {
	int i, j;

	int sizeImage = atoi(argv[2]);
	int nrays     = atoi(argv[3]);
	int nangles   = atoi(argv[4]);

	FILE *fp=fopen(argv[1], "r");

	float *image;
	float *sino;

	image = (float *)malloc(sizeImage*sizeImage*sizeof(float));
	sino = (float *)malloc(nangles*nrays*sizeof(float));
	for (i = 0; i < nangles*nrays; i++)
		fscanf(fp, "%f", &sino[i]);

	BackWithTexture(image, sino, sizeImage, nrays, nangles);

	for(i=0; i< sizeImage; i++) {
		for(j=0; j< sizeImage; j++) {
			fprintf(stdout, "%f ", image[sizeImage*(sizeImage-1-i) + j]);
		}
	    	fprintf(stdout, "\n");
	    }

	free(image);
	free(sino);
	fclose(fp);

}
