#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define PI 3.141592

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }




__global__ void raysum(float *dev_f , float *dev_r , int wdF, int wdR, float dtheta, float dt, int nrays){

	float ini, delta, x, y, cumsum, tol, ctheta, stheta, ttheta, theta, t;
	int X, Y, i, j;

	i = blockDim.x * blockIdx.x + threadIdx.x;
	j = blockDim.y * blockIdx.y + threadIdx.y;

	if ((i<wdR) && (j < nrays) ){

	theta = i*dtheta;
	t = -1.0 + j*dt;

	tol = 1.0/sqrtf(2);
	ini = -tol;
	delta = (float) sqrtf(2)/(wdF-1);

	ctheta = cosf(theta);
	stheta = sinf(theta);
	ttheta = tanf(theta);

	if(stheta < tol){
		cumsum = 0;
	for(Y = 0; Y < wdF; Y++){
		y = ini + Y*delta;
		x = (t/ctheta - y*ttheta);
		X = (int) floorf((x - ini)/delta);
	  	if(X > -1 && X < wdF-1){
			cumsum += (dev_f[Y*wdF + (X+1)] - dev_f[Y*wdF + X])*(x - (ini + X*delta))/delta + dev_f[Y*wdF + X];

		}
	}
	dev_r[j*wdR + i] = cumsum/fabsf(ctheta);
	}
	else{
	cumsum = 0;
	for(X = 0; X < wdF; X++){
		x = ini + X*delta;
		y = (t/stheta - x/ttheta);
		Y = (int) floorf((y - ini)/delta);
		if(Y > -1 && Y < wdF-1){
			cumsum += (dev_f[(Y+1)*wdF + X] - dev_f[Y*wdF + X])*(y - (ini + Y*delta))/delta + dev_f[Y*wdF + X];

		}
	}
	dev_r[j*wdR + i] = cumsum/fabsf(stheta);
	}
	}
}



int main(int argc, char *argv[]) {
	int i, j;
	float dt, dtheta;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

	int sizeImage = atoi(argv[2]);
	int nrays	  = atoi(argv[3]);
	int nangles	  = atoi(argv[4]);

	int wdf = sizeImage;
	int wdr = nangles;
	FILE *fp = fopen(argv[1], "r");

	float *f;
	float *radon;
	float *dev_r = NULL;
	float *dev_f = NULL;

	unsigned int grid1, grid2;
	grid1 = (unsigned int) ceilf(((float)(nangles)/16));
	grid2 = (unsigned int) ceilf(((float)(nrays)/16));
	fprintf(stderr, "%d %d\n", grid1, grid2);

	dim3 grid(grid1, grid2, 1);
	dim3 blocks(16, 16, 1);

	CUDA_CHECK_RETURN(hipMalloc((void**) &dev_f, sizeof(float)*sizeImage*sizeImage));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_r , nangles*nrays*sizeof(float) ) );

	radon = (float *)malloc(nangles*nrays*sizeof(float));
	f = (float *)malloc(sizeImage*sizeImage*sizeof(float));
	for (i = 0; i < sizeImage*sizeImage; i++)
		fscanf(fp, "%f", &f[i]);

	CUDA_CHECK_RETURN(hipMemcpy (dev_f , f , sizeImage*sizeImage*sizeof(float) , hipMemcpyHostToDevice));

	hipEventRecord(start);

	dt = 2.0/(nrays-1);
	dtheta = PI/(nangles-1);
	raysum<<<grid, blocks>>>(dev_f, dev_r, wdf, wdr, dtheta, dt, nrays);



	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy (radon , dev_r , nangles*nrays*sizeof(float) , hipMemcpyDeviceToHost) );
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	fprintf(stderr, "%f ms\n", milliseconds);

	for ( i = 0; i < nrays ; i++){
		for(j=0 ; j<nangles; j++){
			fprintf(stdout, "%f ", radon[(nrays-1-i)*wdr + (nangles-1-j)]);
		}
		fprintf(stdout, "\n");
	}


	CUDA_CHECK_RETURN(hipFree((void*) dev_r));
	CUDA_CHECK_RETURN(hipFree((void*) dev_f));
	CUDA_CHECK_RETURN(hipDeviceReset());

	free(radon);
	free(f);
	fclose(fp);

	return 0;
}
