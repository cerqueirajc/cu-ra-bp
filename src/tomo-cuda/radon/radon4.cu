#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define PI 3.141592653589793238462643383279502884197169399375105820974944592307816406286
//#define INIC -0.7071067811865475244008
#define INIC -1.0

#define TPBXr 16
#define TPBYr 16


texture<float, hipTextureType2D, hipReadModeElementType> texImage;


__global__ void radon_kernel(float* output, float dt, float dtheta, int sizeImage, int nrays, int nangles, float delta, float idelta)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
     int j = 2*(blockIdx.y * blockDim.y + threadIdx.y);

	float cond;
	float ini, x, y, cumsum1, cumsum2, ctheta, stheta, ttheta, theta, t, X, Y;

    if((i < nangles) && (j < nrays))
    {
    	theta = PI - i*dtheta;

    	ini = (float)INIC;

	cond = sqrtf(2)/2;

    	ctheta =cosf(theta);
    	stheta =sinf(theta);
    	ttheta =tanf(theta);

    	if(stheta < cond){
    		cumsum1 = 0;
    		cumsum2 = 0;
    	for(Y = 0; Y < sizeImage; Y++){
				t = -1.0 + j*dt;	
				y = ini + Y*delta;
				x = (t/ctheta - y*ttheta);
				X = (float)((x - ini)*idelta);
				t = -1.0 + (j+1)*dt;
				//if(X >= 0 && X <= (sizeImage-1))
					cumsum1 += tex2D(texImage, X + 0.5f, Y + 0.5f);
				////////////////////////////
				//t = -1.0 + (j+1)*dt;
				x = (t/ctheta - y*ttheta);
				X = (float)((x - ini)*idelta);
				//if(X >= 0 && X < (sizeImage-1))
					cumsum2 += tex2D(texImage, X + 0.5f, Y + 0.5f);
    		}
    		output[(j)*nangles + (i)] = delta*cumsum1/fabsf(ctheta);
    		output[(j+1)*nangles + (i)] = delta*cumsum2/fabsf(ctheta);
    	}

    	else{
    		cumsum1 = 0;
    		cumsum2 = 0;
    	for(X = 0; X < sizeImage; X++){
    			x = ini + X*delta;

    			t = -1.0 + j*dt;
    			y = (t/stheta - x/ttheta);
    			Y = (float)((y - ini)*idelta);
    			//if(Y >= 0 && Y <= (sizeImage-1))
    				cumsum1 += tex2D(texImage, X + 0.5f, Y + 0.5f);
    			/////////////////////
    			t = -1.0 + (j+1)*dt;
    			y = (t/stheta - x/ttheta);
    			Y = (float)((y - ini)*idelta);
    			//if(Y >= 0 && Y <= (sizeImage-1))
    				cumsum2 += tex2D(texImage, X + 0.5f, Y + 0.5f);
    		}
    		output[(j)*nangles + (i)] = delta*cumsum1/fabsf(stheta);
    		output[(j+1)*nangles + (i)] = delta*cumsum2/fabsf(stheta);
    	}

    }
}



void RadonWithTexture(float* h_output, float* h_input, int sizeImage, int nrays, int nangles)
{
    float* d_output;
    float dt = 2.0/(nrays-1);
    float dtheta = PI/(nangles);
    int size = sizeImage*sizeImage*sizeof(float);
    float delta = (float) 2*fabsf(INIC)/(sizeImage-1);
    float idelta = 1/delta;
	//delta = (float) sqrtf(2)/(sizeImage-1);



    /////// KERNEL EXECUTION TIME TEST
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			float milliseconds2 = 0;

			hipEventRecord(start);
	//////////////////////////////////

    // Allocate CUDA array in device memory (phantom matrix)
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, sizeImage, sizeImage);

    // Copy to device memory the phantom matrix
    hipMemcpyToArray(cuArray, 0, 0, h_input, size , hipMemcpyHostToDevice);

    // Set texture parameters
    texImage.addressMode[0] = hipAddressModeBorder;
    texImage.addressMode[1] = hipAddressModeBorder;
    texImage.filterMode     = hipFilterModeLinear;
    /*texImage.normalized     = true;*/

    // Bind the array to the texture reference
    hipBindTextureToArray(texImage, cuArray, channelDesc);

    // Allocate GPU buffers for the output image
    hipMalloc(&d_output, sizeof(float) * nrays * nangles);

    // GRID and BLOCKS SIZE
    dim3 threadsPerBlock(TPBXr,TPBYr);
    dim3 grid((nangles/threadsPerBlock.x) + 1, (nrays/threadsPerBlock.y)/2 + 1);

    //KERNEL EXECUTION
    radon_kernel<<<grid, threadsPerBlock>>>(d_output, dt, dtheta, sizeImage, nrays, nangles, delta, idelta);
    hipDeviceSynchronize();

    /////// PRINT KERNEL EXECUTION TIME
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&milliseconds2, start, stop);
			fprintf(stderr, "%f ms\n", milliseconds2);
    ///////////////////////////////////

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(h_output, d_output, sizeof(float) * nrays * nangles, hipMemcpyDeviceToHost);

    // unbind texture from buffer
    hipUnbindTexture(texImage);
    hipFreeArray(cuArray);
    hipFree(d_output);
    hipDeviceReset();
}


int main(int argc, char *argv[]) {
	int i, j;

	FILE *fp = fopen(argv[1], "r");
	int sizeImage = atoi(argv[2]);
	int nrays	  = atoi(argv[3]);
	int nangles	  = atoi(argv[4]);

	float *f;
	float *radon;

	radon = (float *)malloc(nangles*nrays*sizeof(float));
	f = (float *)malloc(sizeImage*sizeImage*sizeof(float));
	for (i = 0; i < sizeImage*sizeImage; i++)
		(void)fscanf(fp, "%f", &f[i]);


	RadonWithTexture(radon, f, sizeImage, nrays, nangles);


	for ( i = 0; i < nrays ; i++){
		for(j=0 ; j<nangles; j++){
			fprintf(stdout, "%f ", radon[(i)*nangles + (j)]);
		}
		fprintf(stdout, "\n");
	}

		free(radon);
		free(f);
		fclose(fp);

		return 0;
}
